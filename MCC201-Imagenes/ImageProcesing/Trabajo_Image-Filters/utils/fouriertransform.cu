#include "hip/hip_runtime.h"
#include "cuda_methods.h"

#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <hipfft/hipfft.h>
#include <stdlib.h>

__global__ void normalize(hipfftComplex* src, unsigned char* dst, int w, int h)
{
    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    if (ii < h*w)
    {
        const int luma32 = sqrt((src[ii].x)*(src[ii].x)+(src[ii].y)*(src[ii].y))/(1.0f*w);
        dst[ii] = luma32 > 255 ? 255 : luma32 < 0 ? 0 : luma32;
    }
}

__global__ void swapRows(hipfftComplex* src, int w, int h)
{
    hipfftComplex *tmp;
    tmp = (hipfftComplex *)malloc(sizeof(hipfftComplex)*w/2);

    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    if (ii < h/2)
    {
        hipfftComplex* i_start = src+ii*w;
        hipfftComplex* i_end = src + (ii+h/2)*w + w/2;
        memcpy(tmp, i_start, sizeof(hipfftComplex)*w/2);
        memcpy(i_start, i_end, sizeof(hipfftComplex)*w/2);
        memcpy(i_end, tmp, sizeof(hipfftComplex)*w/2);
    }else if (ii < h) {
        hipfftComplex* i_start = src+(ii-h/2)*w +w/2;
        hipfftComplex* i_end = src + ii*w;
        memcpy(tmp, i_start, sizeof(hipfftComplex)*w/2);
        memcpy(i_start, i_end, sizeof(hipfftComplex)*w/2);
        memcpy(i_end, tmp, sizeof(hipfftComplex)*w/2);
    }

    free(tmp);
}

__global__ void unnormalize(unsigned char* src, hipfftComplex* dst, int w, int h)
{
    int ii = blockDim.x * blockIdx.x + threadIdx.x;
    if (ii < h*w)
    {
        dst[ii].x = (float)src[ii];
        dst[ii].y = 0;
    }
}

void img2fft(unsigned char *src, unsigned char *dst, int w, int h){
    hipfftHandle plan;
    hipfftComplex *src_d;
    hipfftComplex *dst_d;
    unsigned char* uchar_d;

    checkCudaErrors(hipMalloc((void**)&src_d, sizeof(hipfftComplex)*w*h));
    checkCudaErrors(hipMalloc((void**)&dst_d, sizeof(hipfftComplex)*w*h));
    checkCudaErrors(hipMalloc((void**)&uchar_d, sizeof(unsigned char)*w*h));

    checkCudaErrors(hipMemcpy(uchar_d, src, h*w *sizeof(unsigned char), hipMemcpyHostToDevice));

    int threadsPerBlock = 32;
    int blocksPerGrid   = (h*w + threadsPerBlock - 1) / threadsPerBlock;
    unnormalize<<<blocksPerGrid,threadsPerBlock>>>(uchar_d, src_d, w, h);

    hipfftPlan2d(&plan, h, w, HIPFFT_C2C);
    hipfftExecC2C(plan, src_d, dst_d, HIPFFT_FORWARD);

    blocksPerGrid   = (h + threadsPerBlock - 1) / threadsPerBlock;
    swapRows<<<blocksPerGrid,threadsPerBlock>>>(dst_d, w, h);

    blocksPerGrid   = (h*w + threadsPerBlock - 1) / threadsPerBlock;
    normalize<<<blocksPerGrid,threadsPerBlock>>>(dst_d, uchar_d, w, h);

    checkCudaErrors(hipMemcpy(dst, uchar_d, h*w *sizeof(unsigned char), hipMemcpyDeviceToHost));

    hipfftDestroy(plan);
    hipFree(src_d);
    hipFree(dst_d);
    hipFree(uchar_d);
}
