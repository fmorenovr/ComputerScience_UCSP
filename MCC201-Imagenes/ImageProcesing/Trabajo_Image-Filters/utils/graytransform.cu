#include "hip/hip_runtime.h"
#include "cuda_methods.h"

#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void rgb2grayKernel(unsigned char *imgr,unsigned char *imgg,unsigned char *imgb,unsigned char *img_gray, int n) {

    int r, g, b;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        r = imgr[index];
        g = imgg[index];
        b = imgb[index];

        img_gray[index] = (unsigned char)( 0.299*r + 0.587*g +  0.114*b);
    }
}

void rgb2gray(unsigned char *imgr, unsigned char *imgg, unsigned char *imgb, unsigned char *img_gray, int n){
    unsigned char *imgr_cuda, *imgg_cuda, *imgb_cuda;
    unsigned char *img_gray_cuda;

    unsigned int nBytes = sizeof(unsigned char) * n;
    int threadsPerBlock = 256;
    int blocksPerGrid   = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipMalloc((void **)&imgr_cuda, nBytes);
    hipMalloc((void **)&imgg_cuda, nBytes);
    hipMalloc((void **)&imgb_cuda, nBytes);
    hipMalloc((void **)&img_gray_cuda, nBytes);

    hipMemcpy(imgr_cuda, imgr, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(imgg_cuda, imgg, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(imgb_cuda, imgb, nBytes, hipMemcpyHostToDevice);

    rgb2grayKernel<<<blocksPerGrid,threadsPerBlock>>>(imgr_cuda, imgg_cuda, imgb_cuda, img_gray_cuda, n);

    hipMemcpy(img_gray, img_gray_cuda, nBytes, hipMemcpyDeviceToHost);

    hipFree(imgr_cuda);
    hipFree(imgg_cuda);
    hipFree(imgb_cuda);
    hipFree(img_gray_cuda);
}
