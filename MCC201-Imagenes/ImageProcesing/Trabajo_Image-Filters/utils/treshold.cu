#include "hip/hip_runtime.h"
#include "cuda_methods.h"

#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

__global__ void rgb2binaryKernel(unsigned char *imgr,unsigned char *imgg,unsigned char *imgb,unsigned char *img_binary, int n, int umbral) {

    int r, g, b;
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    if (index < n){
        r = imgr[index];
        g = imgg[index];
        b = imgb[index];

        img_binary[index] = (unsigned char)( 0.299*r + 0.587*g +  0.114*b)>umbral?255:0;
    }
}

void rgb2binary(unsigned char *imgr, unsigned char *imgg, unsigned char *imgb, unsigned char *img_binary, int n, int umbral){
    unsigned char *imgr_cuda, *imgg_cuda, *imgb_cuda;
    unsigned char *img_binary_cuda;

    unsigned int nBytes = sizeof(unsigned char) * n;
    int threadsPerBlock = 256;
    int blocksPerGrid   = (n + threadsPerBlock - 1) / threadsPerBlock;

    hipMalloc((void **)&imgr_cuda, nBytes);
    hipMalloc((void **)&imgg_cuda, nBytes);
    hipMalloc((void **)&imgb_cuda, nBytes);
    hipMalloc((void **)&img_binary_cuda, nBytes);

    hipMemcpy(imgr_cuda, imgr, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(imgg_cuda, imgg, nBytes, hipMemcpyHostToDevice);
    hipMemcpy(imgb_cuda, imgb, nBytes, hipMemcpyHostToDevice);

    rgb2binaryKernel<<<blocksPerGrid,threadsPerBlock>>>(imgr_cuda, imgg_cuda, imgb_cuda, img_binary_cuda, n, umbral);

    hipMemcpy(img_binary, img_binary_cuda, nBytes, hipMemcpyDeviceToHost);

    hipFree(imgr_cuda);
    hipFree(imgg_cuda);
    hipFree(imgb_cuda);
    hipFree(img_binary_cuda);
}
