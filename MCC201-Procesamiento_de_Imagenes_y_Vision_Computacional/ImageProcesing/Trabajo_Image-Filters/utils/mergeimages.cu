#include "hip/hip_runtime.h"
#include "cuda_methods.h"

#include <>
// CUDA runtime
#include <hip/hip_runtime.h>

// helper functions and utilities to work with CUDA
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>

typedef unsigned char uchar;

__global__ void addImageKernel(uchar *imgr, uchar *imgg, uchar *imgb,
                               uchar *imgr_k, uchar *imgg_k, uchar *imgb_k,
                               int w, int h, float index)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    if(idx < w && idy <h){
        imgr[idy*w + idx] = (1-index)*imgr[idy*w + idx] + index*imgr_k[idy*w + idx];
        imgg[idy*w + idx] = (1-index)*imgg[idy*w + idx] + index*imgg_k[idy*w + idx];
        imgb[idy*w + idx] = (1-index)*imgb[idy*w + idx] + index*imgb_k[idy*w + idx];
    }
}

void addImage(uchar *imgr, uchar *imgg, uchar *imgb,
              uchar *imgr_k, uchar *imgg_k, uchar *imgb_k,
              int w, int h, float index)
{
    uchar *d_imgr, *d_imgg, *d_imgb;
    uchar *d_imgr_k, *d_imgg_k, *d_imgb_k;

    checkCudaErrors(hipMalloc((void **)&d_imgr, sizeof(uchar)*w*h));
    checkCudaErrors(hipMalloc((void **)&d_imgg, sizeof(uchar)*w*h));
    checkCudaErrors(hipMalloc((void **)&d_imgb, sizeof(uchar)*w*h));

    checkCudaErrors(hipMalloc((void **)&d_imgr_k, sizeof(uchar)*w*h));
    checkCudaErrors(hipMalloc((void **)&d_imgg_k, sizeof(uchar)*w*h));
    checkCudaErrors(hipMalloc((void **)&d_imgb_k, sizeof(uchar)*w*h));

    checkCudaErrors(hipMemcpy(d_imgr, imgr, sizeof(uchar)*w*h, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_imgg, imgg, sizeof(uchar)*w*h, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_imgb, imgb, sizeof(uchar)*w*h, hipMemcpyHostToDevice));

    checkCudaErrors(hipMemcpy(d_imgr_k, imgr_k, sizeof(uchar)*w*h, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_imgg_k, imgg_k, sizeof(uchar)*w*h, hipMemcpyHostToDevice));
    checkCudaErrors(hipMemcpy(d_imgb_k, imgb_k, sizeof(uchar)*w*h, hipMemcpyHostToDevice));

    dim3 threadsPerBlock(8, 8);
    dim3 blocksPerGrid((w + 7)/8, (h + 7)/8);
    addImageKernel<<<blocksPerGrid,threadsPerBlock>>>(d_imgr, d_imgg, d_imgb,
                                                      d_imgr_k, d_imgg_k, d_imgb_k,
                                                      w, h, index);

    checkCudaErrors(hipMemcpy(imgr, d_imgr, sizeof(uchar)*w*h, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(imgg, d_imgg, sizeof(uchar)*w*h, hipMemcpyDeviceToHost));
    checkCudaErrors(hipMemcpy(imgb, d_imgb, sizeof(uchar)*w*h, hipMemcpyDeviceToHost));

    checkCudaErrors(hipFree(d_imgr));
    checkCudaErrors(hipFree(d_imgg));
    checkCudaErrors(hipFree(d_imgb));

    checkCudaErrors(hipFree(d_imgr_k));
    checkCudaErrors(hipFree(d_imgg_k));
    checkCudaErrors(hipFree(d_imgb_k));
}
